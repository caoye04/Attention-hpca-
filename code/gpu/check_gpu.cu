#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("GPU Name: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    return 0;
}